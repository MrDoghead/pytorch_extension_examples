
#include <hip/hip_runtime.h>
extern "C" {
__global__ void matmul_kernel(const float* pfMatrixA, const float* pfMatrixB, float* pfMatrixC, int m, int k, int n)
{
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    for(int i =0; i < k; i++)
    {
        sum += pfMatrixA[nRow * k + i] * pfMatrixB[i * n + nCol];
    }
    pfMatrixC[nRow * n + nCol] = sum;
}
}
